#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <ctime>
#include <stdio.h>
#include "stdlib.h"

#include "windows.h"
#include "GPUMatrix.h"
#include <math.h>

#define V2D(i, j) (i) * size + (j)
#define BLOCK_SIZE 256

hipError_t launchCuda(int *c, const int *a, const int *b, size_t size);
void algorithm(int *c, const int *a, const int *b, size_t size);


// Globals

int initTime = 0;
bool g_Bl1_GPU = true;
bool g_Bl2_GPU = true;
bool g_Bl3_GPU = true;


int (*pInverse)(float*, float*, int, int);

/////////////////////////////////////////////////////////////////////////////////////////////
//									 CPU KERNELS										   //
/////////////////////////////////////////////////////////////////////////////////////////////

__host__ void Bl1_CPU(float *X_new, float *X_old, float *A, int j, int size)
{
    for (int i = j - 1; i < size; ++i)
	{
		X_new[i] = X_new[i] - A[i * size + j]*X_old[j];
	}
}

__host__ void Bl2_CPU(float *X_new, float *A, int t, int size)
{
	for (int j = max(1, t - size); j < (t - 1)/2; ++j)
	{
		int i = t - j - 1;
		X_new[i] = X_new[i] - A[i * size + j]*X_new[j];
	}
}

void helpBl_CPU(float *X, float *B)
{
	int i = 0;// threadIdx.x;
	X[i] = B[i];
}

/////////////////////////////////////////////////////////////////////////////////////////////
//									CUDA KERNELS										   //
/////////////////////////////////////////////////////////////////////////////////////////////

__global__ void Bl1(float *X_new, float *X_old, float *A, int _j, int size, int stride)
{
	int j = blockIdx.y * blockDim.y + _j;
    int i = blockIdx.x * blockDim.x + (_j - 1) + threadIdx.x;
    X_new[i] = X_new[i] - A[i * stride + j]*X_old[j];
}

__global__ void Bl2(float *X_new, float *A, int t, int size, int stride)
{
	int j = ((t - size) < 1)? 1: (t - size) + threadIdx.x;

	int i = blockIdx.x * blockDim.x  + t - j;
	j += blockIdx.y * blockDim.y;

	X_new[i] = X_new[i] - A[i * stride + j]*X_new[j];
}

__global__ void helpBl(float *X, float *B)
{
	int i = threadIdx.x;
	X[i] = B[i];
}

/////////////////////////////////////////////////////////////////////////////////////////////
//							CUDA MATRIX-VECTOR MULTIPLICATION							   //
/////////////////////////////////////////////////////////////////////////////////////////////
__global__ void MVKernel_gm(float* A, float* X, float* Y, int size)
{
	int bx = blockIdx.x; 
	int tx = threadIdx.x; 
	int Row = bx * blockDim.x + tx;
	float Pvalue = 0;
   
	for (unsigned int k = 0; k < size; k++) 
		{
			if(Row < size)         
			Pvalue += A[Row * size + k] * X[k];		
		}
	
	__syncthreads();
	
	if(Row < size)  		
	  Y[Row] = Pvalue;
	__syncthreads();
}

/////////////////////////////////////////////////////////////////////////////////////////////
//									CUDA ZEIDEL ALGO									   //
/////////////////////////////////////////////////////////////////////////////////////////////
/// r - number of blocks
/// K - number of iterations
/// A, B, X - parts of linear system
/// size - size of system
hipError_t algorithm(const int &r, const int &K, float *A, float *A_inv, float *B, const size_t &size, float *X)
{
	hipError_t cudaStatus;
	float *X_new;
	float *X_old;

	float *test = new float[size];
	float *test1 = new float[size];

	hipMalloc((void**)&X_new, size * sizeof(float));
	hipMalloc((void**)&X_old, size * sizeof(float));

	hipMemcpy(X_old, B, size * sizeof(float), hipMemcpyDeviceToDevice);


	// Additional memory for CPU
	float *h_X_new = (float*)malloc(size * sizeof(float));
	float *h_X_old = (float*)malloc(size * sizeof(float));
	float *h_A = (float*)malloc(size * size * sizeof(float));
	hipMemcpy(h_X_new, X_new, size * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_X_old, X_old, size * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_A, A, size * size * sizeof(float), hipMemcpyDeviceToHost);

	for (int i = 0; i < K; ++i)
	{
		printf("%i", i);

		hipMemcpy(X_new, B, size * sizeof(float), hipMemcpyDeviceToDevice);

		hipMemcpy(h_X_new, X_new, size * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(h_X_old, X_old, size * sizeof(float), hipMemcpyDeviceToHost);

		int n = size / ((g_Bl1_GPU)?r:1);

		for (int j = 1; j < n; ++j)
		{
			if (g_Bl1_GPU)
			{
				int num = n - j;
				if (num > 1024)
					fprintf(stderr, "\n\t !Number of thread in block is greater than 1024 (Block1)!\n");
				dim3 numBlocks(r, r);
				Bl1<<<numBlocks, num>>>(X_new, X_old, A, j, n, size);
				cudaStatus = hipDeviceSynchronize();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "\n !hipDeviceSynchronize returned error code %d after launching Block1!\n");
					return cudaStatus;
				}
			}
			else
			{
				Bl1_CPU(h_X_new, h_X_old, h_A, j, size);
			}
		}

		if (!g_Bl1_GPU)
			hipMemcpy(X_new, h_X_new, size * sizeof(float), hipMemcpyHostToDevice);

		n = size / ((g_Bl2_GPU)?r:1);

		for (int t = 2; t < 2*n; ++t)
		{
			if (g_Bl2_GPU)
			{
				int num = ((t - 1) / 2) - max(1, t - n) + 1;
				if (t < 3 || t == 2*n - 1)
					continue;
				if (num > 1024)
					fprintf(stderr, "\n\t !Number of thread in block is greater than 1024(Block2)!\n");
				if (num < 0)
					fprintf(stderr, "\n\t !Number of thread in block is less than 1(Block2)! t = %d\n", t);

				dim3 numBlocks(r, r);
				Bl2<<<numBlocks, num>>>(X_new, A, t, n, size);
				cudaStatus = hipDeviceSynchronize();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "\n\t !hipDeviceSynchronize returned error code %d after launching Block2!\n", cudaStatus);
					return cudaStatus;
				}
			}
			else
			{
				Bl2_CPU(h_X_new, h_A, t, size);
			}
			
			if ((t / 2) % 2 == 0)
			{
				// Multiply matrix by vector
				if (!g_Bl2_GPU)
					hipMemcpy(X_new, h_X_new, size * sizeof(float), hipMemcpyHostToDevice);

				dim3 dimGrid((size - 1) / BLOCK_SIZE + 1);
				dim3 dimBlock(BLOCK_SIZE);
				MVKernel_gm<<<dimGrid,dimBlock>>>(A, X_new, X_new, size);

				if (!g_Bl2_GPU)
					hipMemcpy(h_X_new, X_new, size * sizeof(float), hipMemcpyDeviceToHost);
			}
		}

		if (!g_Bl2_GPU)
			hipMemcpy(X_new, h_X_new, size * sizeof(float), hipMemcpyHostToDevice);

		hipMemcpy(X_old, X_new, size * sizeof(float), hipMemcpyDeviceToDevice);
	}
	
	hipMemcpy(X, X_new, size * sizeof(float), hipMemcpyDeviceToDevice);
}

inline float randf(float min, float max)
{
	return min + (float)rand()/((float)RAND_MAX/(max-min));
}

void GenerateEquation(const int &size, float *A, float *B)
{
	float min = -1000.f;
	float max = 1000.f;
	
	srand((unsigned)time(0));
	for(int i = 0; i < size; ++i)
	{
		float absSum = 0.f;
		
		for(int j = 0; j < size; ++j)
		{
			A[i*size + j] = randf(min, max);
			absSum += fabsf(A[i*size + j]);
		}

		absSum -= fabsf(A[i*size + i]);
		A[i*size + i] = absSum + randf(5.f, max);
		if(randf(0.f, 1.f) < 0.5f)
		{
			A[i*size + i] *= -1.f;
		}

		B[i] = randf(min, max);
	}
}

/////////////////////////////////////////////////////////////////////////////////////////////
//									CUDA TESTS  										   //
/////////////////////////////////////////////////////////////////////////////////////////////

void TestFunc()
{
	printf("blah-blah\n");
}
int cudaTest(const int &size, float &time, float *answer)
{
	srand((unsigned)std::time(NULL));
	time = initTime;
	for (int i = 0; i < size; ++i)
		answer[i] = rand();
	return 0;
}
void cudaSetInitTime(int t)
{
	initTime = t;
}

/////////////////////////////////////////////////////////////////////////////////////////////
//									CUDA SETTINGS										   //
/////////////////////////////////////////////////////////////////////////////////////////////

void setProcessingUnit(bool Bl1_GPU, bool Bl2_GPU, bool Bl3_GPU)
{
	g_Bl1_GPU = Bl1_GPU;
	g_Bl2_GPU = Bl2_GPU;
	g_Bl3_GPU = Bl3_GPU;
}

/////////////////////////////////////////////////////////////////////////////////////////////
//									CUDA ROUTIN  										   //
/////////////////////////////////////////////////////////////////////////////////////////////

// Helper function for using CUDA to add vectors in parallel.
hipError_t launchCuda(const int &size, const int &r, float &time, float *answer)
{
    float *dev_a = 0;
	float *dev_a_inv = 0;
    float *dev_b = 0;
    float *dev_x = 0;

	float *A = (float*)(malloc(size * size * sizeof(float)));
	float *A_inv = (float*)(malloc(size * size * sizeof(float)));
	float *B = (float*)(malloc(size * sizeof(float)));

	GenerateEquation(size, A, B);
	GPUInverse(A, A_inv, size);


    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)
    cudaStatus = hipMalloc((void**)&dev_a, size * size * sizeof(float));
	cudaStatus = hipMalloc((void**)&dev_a_inv, size * size * sizeof(float));
	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(float));
    cudaStatus = hipMalloc((void**)&dev_x, size * sizeof(float));
	
    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, A, size * size * sizeof(float),	hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(dev_a_inv, A_inv, size * size * sizeof(float),	hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(dev_b, B, size * sizeof(float),			hipMemcpyHostToDevice);

    // Launch algorithm
	int K = 10;
	printf("start\n");

	__int64 startTime;
	__int64 endTime;
	QueryPerformanceCounter((LARGE_INTEGER*)&startTime);

    cudaStatus = algorithm(r, K, dev_a, dev_a_inv, dev_b, size, dev_x);

	QueryPerformanceCounter((LARGE_INTEGER*)&endTime);
	printf("stop\n");

	__int64 countsPerSec;
	double secPerCount;
	QueryPerformanceFrequency((LARGE_INTEGER*)&countsPerSec);
	secPerCount = 1.0 / (double)countsPerSec;

	time = (float)((endTime - startTime) * secPerCount);

    // Copy output vector from GPU buffer to host memory.
    hipMemcpy(answer, dev_x, size * sizeof(float), hipMemcpyDeviceToHost);

Error:
	free(A);
	free(A_inv);
	free(B);
    hipFree(dev_x);
	hipFree(dev_a_inv);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}

int cudaMain(const int &size, const int& r, float &time, float *answer)
{
    hipError_t cudaStatus = launchCuda(size, r, time, answer);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Seidel with cuda failed!\n");
        return 1;
    }
	
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

