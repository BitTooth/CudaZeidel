#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <ctime>
#include <stdio.h>
#include "stdlib.h"

#include "windows.h"

#define V2D(i, j) (i) * size + (j)

hipError_t launchCuda(int *c, const int *a, const int *b, size_t size);
void algorithm(int *c, const int *a, const int *b, size_t size);


// Globals

int initTime = 0;
bool g_Bl1_GPU = true;
bool g_Bl2_GPU = true;
bool g_Bl3_GPU = true;

/////////////////////////////////////////////////////////////////////////////////////////////
//									 CPU KERNELS										   //
/////////////////////////////////////////////////////////////////////////////////////////////

__host__ void Bl1_CPU(float *X_new, float *X_old, float *A, int j, int size)
{
    for (int i = j - 1; i < size; ++i)
	{
		X_new[i] = X_new[i] - A[i * size + j]*X_old[j];
	}
}

__host__ void Bl2_CPU(float *X_new, float *A, int t, int size)
{
	for (int j = max(1, t - size); j < (t - 1)/2; ++j)
	{
		int i = t - j - 1;
		X_new[i] = X_new[i] - A[i * size + j]*X_new[j];
	}
}

void helpBl_CPU(float *X, float *B)
{
	int i = 0;// threadIdx.x;
	X[i] = B[i];
}

/////////////////////////////////////////////////////////////////////////////////////////////
//									CUDA KERNELS										   //
/////////////////////////////////////////////////////////////////////////////////////////////

__global__ void Bl1(float *X_new, float *X_old, float *A, int _j, int size, int stride)
{
	int j = blockIdx.y * blockDim.y + _j;
    int i = blockIdx.x * blockDim.x + (_j - 1) + threadIdx.x;
    X_new[i] = X_new[i] - A[i * stride + j]*X_old[j];
}

__global__ void Bl2(float *X_new, float *A, int t, int size, int stride)
{
	int j = ((t - size) < 1)? 1: (t - size) + threadIdx.x;

	int i = blockIdx.x * blockDim.x  + t - j;
	j += blockIdx.y * blockDim.y;

	X_new[i] = X_new[i] - A[i * stride + j]*X_new[j];
}

__global__ void helpBl(float *X, float *B)
{
	int i = threadIdx.x;
	X[i] = B[i];
}

/////////////////////////////////////////////////////////////////////////////////////////////
//									CUDA ZEIDEL ALGO									   //
/////////////////////////////////////////////////////////////////////////////////////////////
/// r - number of blocks
/// K - number of iterations
/// A, B, X - parts of linear system
/// size - size of system
hipError_t algorithm(const int &r, const int &K, float *A, float *B, const size_t &size, float *X)
{
	hipError_t cudaStatus;
	float *X_new;
	float *X_old;

	float *test = new float[size];
	float *test1 = new float[size];

	hipMalloc((void**)&X_new, size * sizeof(float));
	hipMalloc((void**)&X_old, size * sizeof(float));

	hipMemcpy(X_old, B, size * sizeof(float), hipMemcpyDeviceToDevice);


	// Additional memory for CPU
	float *h_X_new = (float*)malloc(size * sizeof(float));
	float *h_X_old = (float*)malloc(size * sizeof(float));
	float *h_A = (float*)malloc(size * size * sizeof(float));
	hipMemcpy(h_X_new, X_new, size * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_X_old, X_old, size * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_A, A, size * size * sizeof(float), hipMemcpyDeviceToHost);

	for (int i = 0; i < K; ++i)
	{
		printf("%i", i);

		hipMemcpy(X_new, B, size * sizeof(float), hipMemcpyDeviceToDevice);

		hipMemcpy(h_X_new, X_new, size * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(h_X_old, X_old, size * sizeof(float), hipMemcpyDeviceToHost);

		int n = size / ((g_Bl1_GPU)?r:1);

		for (int j = 1; j < n; ++j)
		{
			if (g_Bl1_GPU)
			{
				int num = n - j;
				if (num > 1024)
					fprintf(stderr, "\n\t !Number of thread in block is greater than 1024 (Block1)!\n");
				dim3 numBlocks(r, r);
				Bl1<<<numBlocks, num>>>(X_new, X_old, A, j, n, size);
				cudaStatus = hipDeviceSynchronize();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "\n !hipDeviceSynchronize returned error code %d after launching Block1!\n");
					return cudaStatus;
				}
			}
			else
			{
				Bl1_CPU(h_X_new, h_X_old, h_A, j, size);
			}
		}

		if (!g_Bl1_GPU)
			hipMemcpy(X_new, h_X_new, size * sizeof(float), hipMemcpyHostToDevice);

		n = size / ((g_Bl2_GPU)?r:1);

		for (int t = 2; t < 2*n; ++t)
		{
			if (g_Bl2_GPU)
			{
				int num = ((t - 1) / 2) - max(1, t - n) + 1;
				if (t < 3 || t == 2*n)
					continue;
				if (num > 1024)
					fprintf(stderr, "\n\t !Number of thread in block is greater than 1024(Block2)!\n");
				if (num < 0)
					fprintf(stderr, "\n\t !Number of thread in block is less than 1(Block2)! t = %d\n", t);

				dim3 numBlocks(r, r);
				Bl2<<<numBlocks, num>>>(X_new, A, t, n, size);
				cudaStatus = hipDeviceSynchronize();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "\n\t !hipDeviceSynchronize returned error code %d after launching Block2!\n", cudaStatus);
					return cudaStatus;
				}
			}
			else
			{
				Bl2_CPU(h_X_new, h_A, t, size);
			}
			
			if ((t / 2) % 2 == 0)
			{
				// Multiply matrix by vector
			}
		}

		if (!g_Bl2_GPU)
			hipMemcpy(X_new, h_X_new, size * sizeof(float), hipMemcpyHostToDevice);

		hipMemcpy(X_old, X_new, size * sizeof(float), hipMemcpyDeviceToDevice);
	}
	
	hipMemcpy(X, X_new, size * sizeof(float), hipMemcpyDeviceToDevice);
}

void GenerateEquation(const int &size, float *A, float *B)
{
	for (int i = 0; i < size; ++i)
	{
		for (int j = 0; j < size; ++j)
		{
			float random = ((i != j)? RAND_MAX * 100 : 100);
			A[i * size + j] = (float)rand() / random;
		}

		B[i] = (float)rand() / (float)(RAND_MAX / size);
	}
}

/////////////////////////////////////////////////////////////////////////////////////////////
//									CUDA TESTS  										   //
/////////////////////////////////////////////////////////////////////////////////////////////

void TestFunc()
{
	printf("blah-blah\n");
}
int cudaTest(const int &size, float &time, float *answer)
{
	srand((unsigned)std::time(NULL));
	time = initTime;
	for (int i = 0; i < size; ++i)
		answer[i] = rand();
	return 0;
}
void cudaSetInitTime(int t)
{
	initTime = t;
}

/////////////////////////////////////////////////////////////////////////////////////////////
//									CUDA SETTINGS										   //
/////////////////////////////////////////////////////////////////////////////////////////////

void setProcessingUnit(bool Bl1_GPU, bool Bl2_GPU, bool Bl3_GPU)
{
	g_Bl1_GPU = Bl1_GPU;
	g_Bl2_GPU = Bl2_GPU;
	g_Bl3_GPU = Bl3_GPU;
}

/////////////////////////////////////////////////////////////////////////////////////////////
//									CUDA ROUTIN  										   //
/////////////////////////////////////////////////////////////////////////////////////////////

// Helper function for using CUDA to add vectors in parallel.
hipError_t launchCuda(const int &size, const int &r, float &time, float *answer)
{
    float *dev_a = 0;
    float *dev_b = 0;
    float *dev_x = 0;

	float *A = (float*)(malloc(size * size * sizeof(float)));
	float *B = (float*)(malloc(size * sizeof(float)));

    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)
    cudaStatus = hipMalloc((void**)&dev_a, size * size * sizeof(float));
	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(float));
    cudaStatus = hipMalloc((void**)&dev_x, size * sizeof(float));


	// Generate matrix for algorithm
	GenerateEquation(size, A, B);

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, A, size * size * sizeof(float),	hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(dev_b, B, size * sizeof(float),			hipMemcpyHostToDevice);

    // Launch algorithm
	int K = 10;
	printf("start\n");

	__int64 startTime;
	__int64 endTime;
	QueryPerformanceCounter((LARGE_INTEGER*)&startTime);

    cudaStatus = algorithm(r, K, dev_a, dev_b, size, dev_x);

	QueryPerformanceCounter((LARGE_INTEGER*)&endTime);
	printf("stop\n");

	__int64 countsPerSec;
	double secPerCount;
	QueryPerformanceFrequency((LARGE_INTEGER*)&countsPerSec);
	secPerCount = 1.0 / (double)countsPerSec;

	time = (float)((endTime - startTime) * secPerCount);

    // Copy output vector from GPU buffer to host memory.
    hipMemcpy(answer, dev_x, size * sizeof(float), hipMemcpyDeviceToHost);

Error:
	free(A);
	free(B);
    hipFree(dev_x);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}

int cudaMain(const int &size, const int& r, float &time, float *answer)
{
    hipError_t cudaStatus = launchCuda(size, r, time, answer);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Seidel with cuda failed!\n");
        return 1;
    }
	
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

