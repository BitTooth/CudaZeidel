#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <ctime>
#include <stdio.h>
#include "stdlib.h"

hipError_t launchCuda(int *c, const int *a, const int *b, size_t size);
void algorithm(int *c, const int *a, const int *b, size_t size);

int initTime = 0;

/////////////////////////////////////////////////////////////////////////////////////////////
//									CUDA KERNELS										   //
/////////////////////////////////////////////////////////////////////////////////////////////

__global__ void Bl1(float *X_new, float *X_old, float **A, int j)
{
    int i = (j - 1) + threadIdx.x;
    X_new[i] = X_new[i] - A[i][j]*X_old[j];
}

__global__ void Bl2(float *X_new, float **A, int t, int n)
{
	int j = fmax((int)1, (int)(t - n)) + threadIdx.x;
	int i = t - j;
	X_new[i] = X_new[i] - A[i][j]*X_new[j];
}

__global__ void helpBl(float *X, float *B)
{
	int i = threadIdx.x;
	X[i] = B[i];
}

/////////////////////////////////////////////////////////////////////////////////////////////
//									CUDA ZEIDEL ALGO									   //
/////////////////////////////////////////////////////////////////////////////////////////////

void algorithm(const int &K, float **A, float *B, const size_t &size, float *X)
{
	float *X_new;
	float *X_old;

	hipMalloc((void**)&X_new, size * sizeof(float));
	hipMalloc((void**)&X_old, size * sizeof(float));

	hipMemcpy(X_old, B, size * sizeof(float), hipMemcpyDeviceToDevice);

	for (int i = 0; i < K; ++i)
	{
		hipMemcpy(X_new, B, size * sizeof(float), hipMemcpyDeviceToDevice);

		for (int j = 2; j < size; ++j)
		{
			int num = n - j;
			Bl1<<<1, num>>>(X_new, X_old, A, j);
		}

		for (int t = 2; t < 2*size; ++t)
		{
			int num = 2 * size - 2;
			Bl2<<<1, num>>>(X_new, A, t, size);
		}

		if ((t / 2) % 2 == 0)
		{
			// Multiply matrix by vector
		}
	}
	
}


/////////////////////////////////////////////////////////////////////////////////////////////
//									CUDA TESTS  										   //
/////////////////////////////////////////////////////////////////////////////////////////////

void TestFunc()
{
	printf("blah-blah\n");
}

int cudaTest(const int &size, float &time, float *answer)
{
	srand((unsigned)std::time(NULL));
	time = initTime;
//	answer = new float[size];
	for (int i = 0; i < size; ++i)
		answer[i] = rand();
	return 0;
}

void cudaSetInitTime(int t)
{
	initTime = t;
}

/////////////////////////////////////////////////////////////////////////////////////////////
//									CUDA ROUTIN  										   //
/////////////////////////////////////////////////////////////////////////////////////////////


int cudaMain(const int &size, float &time, float *answer)
{
    const int arraySize = 5;
    const int *a = new int[size];
    const int *b = new int[size];
    int *c = new int[size];

    // Add vectors in parallel.
    hipError_t cudaStatus = launchCuda(c, a, b, size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

	for (int i = 0; i < size; ++i)
	{
		answer[i] = c[i];
	}

    //printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
    //    c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

bool loadInput(char *path, float **A, float *B, int &size)
{
	return true;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t launchCuda(int *c, const int *a, const int *b, size_t size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    algorithm(dev_c, dev_a, dev_b, size);

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
