#include "hip/hip_runtime.h"
// Kernels/setIdentity_kernel.cu
// ���������� ��������� �������
#include "../GPUMatrix.h"

__global__ void setIdentity (float* matrix, int size)
{ 
	// ��������� ������� �������
	unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

	if ((xIndex >= size) || (yIndex != 0))
		return;

	// ���������� ������� � �������
	unsigned int index = xIndex * 2 * size + size + xIndex;
	matrix[index] = 1; 
}
